#include "hip/hip_runtime.h"
include <iostream>
#include <hip/hip_runtime.h>

const int N = 512;

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // Allocate device memory
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy arrays to device
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N blocks
    add<<<N,1>>>(dev_a, dev_b, dev_c);

    // Copy array c back to host
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Display the result
    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }

    // Cleanup
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
